#include <iostream>
#include <hip/hip_runtime.h>

/**
 * @brief GPU에서 실행될 커널(Kernel) 함수
 * * __global__ 키워드는 이 함수가 CPU(호스트)에 의해 호출되어
 * * GPU(디바이스)에서 실행된다는 것을 나타냅니다.
 */
__global__ void hello_from_gpu() {
    // threadIdx.x는 현재 실행 중인 스레드의 블록 내 ID를 나타냅니다.
    // GPU의 printf는 모든 스레드의 출력을 버퍼에 모았다가
    // 커널 실행이 끝난 후 한 번에 호스트로 전송합니다.
    printf("Hello, World! from GPU thread %d\n", threadIdx.x);
}

int main() {
    std::cout << "Hello from CPU!" << std::endl;

    // GPU에서 hello_from_gpu 커널을 실행합니다.
    // <<<1, 5>>>는 1개의 스레드 블록(block)에 5개의 스레드(thread)를
    // 생성하여 실행하라는 의미입니다.
    hello_from_gpu<<<1, 5>>>();

    // cudaDeviceSynchronize()는 GPU에서 실행된 모든 작업이
    // 완전히 끝날 때까지 CPU가 기다리도록 하는 동기화 함수입니다.
    // 이 함수가 없으면, GPU가 printf를 실행하기 전에
    // main 함수가 먼저 종료될 수 있습니다.
    hipDeviceSynchronize();

    std::cout << "Kernel launch finished." << std::endl;

    return 0;
}
